#include "hip/hip_runtime.h"
#include "tpat_test_conv_with_strides_padding.h"
#include <hip/hip_runtime.h>
#include <thread>
#include <stdio.h>
#include <nvfunctional>
#include <chrono>

#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

using namespace nvinfer1;
using namespace plugin;

// CUDA Runtime error messages
#ifdef __DRIVER_TYPES_H__
static const char *_cudaGetErrorEnum(hipError_t error)
{
  return hipGetErrorName(error);
}
#endif

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line)
{
  if (result)
  {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)



#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = int;
  using uint64_t = unsigned int;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t int
  #define uint64_t unsigned int
#endif
extern "C" __global__ void __launch_bounds__(3) tvmgen_default_fused_nn_conv2d_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[7];
  __shared__ float placeholder_shared[3];
  compute_local[(0)] = 0.000000e+00f;
  pad_temp_shared[((((int)threadIdx.x) * 3))] = ((((1 <= ((int)blockIdx.y)) && (1 <= ((int)threadIdx.x))) && (((int)threadIdx.x) < 2)) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) - 6))] : 0.000000e+00f);
  if (((int)threadIdx.x) < 2) {
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 1))] = ((1 <= ((int)blockIdx.y)) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) - 5))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 2) {
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 2))] = ((1 <= ((int)blockIdx.y)) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) - 4))] : 0.000000e+00f);
  }
  placeholder_shared[(((int)threadIdx.x))] = placeholder1[(((int)threadIdx.x))];
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] * placeholder_shared[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 2))] * placeholder_shared[(2)]));
  __syncthreads();
  pad_temp_shared[((((int)threadIdx.x) * 3))] = (((1 <= ((int)threadIdx.x)) && (((int)threadIdx.x) < 2)) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) - 1))] : 0.000000e+00f);
  if (((int)threadIdx.x) < 2) {
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 1))] = placeholder[(((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)))];
  }
  if (((int)threadIdx.x) < 2) {
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 2))] = placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) + 1))];
  }
  placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((int)threadIdx.x) + 3))];
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] * placeholder_shared[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 2))] * placeholder_shared[(2)]));
  __syncthreads();
  pad_temp_shared[((((int)threadIdx.x) * 3))] = ((((((int)blockIdx.y) < 3) && (1 <= ((int)threadIdx.x))) && (((int)threadIdx.x) < 2)) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) + 4))] : 0.000000e+00f);
  if (((int)threadIdx.x) < 2) {
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 1))] = ((((int)blockIdx.y) < 3) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) + 5))] : 0.000000e+00f);
  }
  if (((int)threadIdx.x) < 2) {
    pad_temp_shared[(((((int)threadIdx.x) * 3) + 2))] = ((((int)blockIdx.y) < 3) ? placeholder[((((((int)blockIdx.y) * 10) + (((int)threadIdx.x) * 3)) + 6))] : 0.000000e+00f);
  }
  placeholder_shared[(((int)threadIdx.x))] = placeholder1[((((int)threadIdx.x) + 6))];
  __syncthreads();
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[((((int)threadIdx.x) * 2))] * placeholder_shared[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 1))] * placeholder_shared[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared[(((((int)threadIdx.x) * 2) + 2))] * placeholder_shared[(2)]));
  compute[(((((int)blockIdx.y) * 3) + ((int)threadIdx.x)))] = compute_local[(0)];
}



PluginFieldCollection tpat_test_conv_with_strides_paddingCreator::mFC{};
std::vector<PluginField> tpat_test_conv_with_strides_paddingCreator::mPluginAttributes;

int tpat_test_conv_with_strides_padding::enqueue(const nvinfer1::PluginTensorDesc* inputDesc, const nvinfer1::PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept {
    
    dim3 dimBlock, dimGrid;
    
    dimGrid = dim3(1,4,1);
    dimBlock = dim3(3,1,1);
    tvmgen_default_fused_nn_conv2d_kernel0<<<dimGrid, dimBlock, 0, stream>>>((float*)inputs[0], (float*)inputs[1], (float*)outputs[0]);
    
}

REGISTER_TENSORRT_PLUGIN(tpat_test_conv_with_strides_paddingCreator);